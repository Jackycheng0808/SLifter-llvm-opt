// %%writefile addMatrices.cu

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void addArrays(int n, float *a, float *b, float *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int n = 1 << 20;  // 1 million elements

    float *h_a = new float[n];
    float *h_b = new float[n];
    float *h_c = new float[n];
    float *d_a, *d_b, *d_c;

    for (int i = 0; i < n; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }

    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));

    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    addArrays<<<numBlocks, blockSize>>>(n, d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print result for verification
    for (int i = 0; i < 10; i++) {
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    std::cout << "COMPLETED SUCCESSFULLY\n";

    return 0;
}